#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuFusedCollisionStream(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const int idx3 = gpuIdxGlobal3(x,y,z);
    
    float fneq[NLINKS];
    float pop[NLINKS];
      
    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        pop[Q] = d.f[idx4];
    }

    float rho_val = 0.0f;
    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) 
        rho_val += pop[Q];

    float inv_rho = 1.0f / rho_val;

    #ifdef D3Q19
        float sum_ux = inv_rho * (pop[1] - pop[2] + pop[7] - pop[8]  + pop[9]  - pop[10] + pop[13] - pop[14] + pop[15] - pop[16]);
        float sum_uy = inv_rho * (pop[3] - pop[4] + pop[7] - pop[8]  + pop[11] - pop[12] + pop[14] - pop[13] + pop[17] - pop[18]);
        float sum_uz = inv_rho * (pop[5] - pop[6] + pop[9] - pop[10] + pop[11] - pop[12] + pop[16] - pop[15] + pop[18] - pop[17]);
    #elif defined(D3Q27)
        float sum_ux = inv_rho * (pop[1] - pop[2] + pop[7] - pop[8]  + pop[9]  - pop[10] + pop[13] - pop[14] + pop[15] - pop[16] + pop[19] - pop[20] + pop[21] - pop[22] + pop[23] - pop[24] + pop[26] - pop[25]);
        float sum_uy = inv_rho * (pop[3] - pop[4] + pop[7] - pop[8]  + pop[11] - pop[12] + pop[14] - pop[13] + pop[17] - pop[18] + pop[19] - pop[20] + pop[21] - pop[22] + pop[24] - pop[23] + pop[25] - pop[26]);
        float sum_uz = inv_rho * (pop[5] - pop[6] + pop[9] - pop[10] + pop[11] - pop[12] + pop[16] - pop[15] + pop[18] - pop[17] + pop[19] - pop[20] + pop[22] - pop[21] + pop[23] - pop[24] + pop[25] - pop[26]);
    #endif

    float ffx_val = d.ffx[idx3];
    float ffy_val = d.ffy[idx3];
    float ffz_val = d.ffz[idx3];

    float fx_corr = ffx_val * 0.5f * inv_rho;
    float fy_corr = ffy_val * 0.5f * inv_rho;
    float fz_corr = ffz_val * 0.5f * inv_rho;

    float ux_val = sum_ux + fx_corr;
    float uy_val = sum_uy + fy_corr;
    float uz_val = sum_uz + fz_corr;

    float uu = 1.5f * (ux_val*ux_val + uy_val*uy_val + uz_val*uz_val);
    float inv_rho_cssq = 3.0f * inv_rho;

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        float pre_feq = gpuComputeEquilibriaSecondOrder(rho_val,ux_val,uy_val,uz_val,uu,Q);
        float he_force = COEFF_HE * pre_feq * ((CIX[Q] - ux_val) * ffx_val +
                                               (CIY[Q] - uy_val) * ffy_val +
                                               (CIZ[Q] - uz_val) * ffz_val) * inv_rho_cssq;
        float feq = pre_feq - he_force; 
        fneq[Q] = pop[Q] - feq;
    }

    float PXX = fneq[1]  + fneq[2]  + fneq[7]  + fneq[8]  + fneq[9]  + fneq[10] + fneq[13] + fneq[14] + fneq[15] + fneq[16];
    float PYY = fneq[3]  + fneq[4]  + fneq[7]  + fneq[8]  + fneq[11] + fneq[12] + fneq[13] + fneq[14] + fneq[17] + fneq[18];
    float PZZ = fneq[5]  + fneq[6]  + fneq[9]  + fneq[10] + fneq[11] + fneq[12] + fneq[15] + fneq[16] + fneq[17] + fneq[18];
    float PXY = fneq[7]  - fneq[13] + fneq[8]  - fneq[14];
    float PXZ = fneq[9]  - fneq[15] + fneq[10] - fneq[16];
    float PYZ = fneq[11] - fneq[17] + fneq[12] - fneq[18];
    #ifdef D3Q27
    PXX += fneq[19] + fneq[20] + fneq[21] + fneq[22] + fneq[23] + fneq[24] + fneq[25] + fneq[26];
    PYY += fneq[19] + fneq[20] + fneq[21] + fneq[22] + fneq[23] + fneq[24] + fneq[25] + fneq[26];
    PZZ += fneq[19] + fneq[20] + fneq[21] + fneq[22] + fneq[23] + fneq[24] + fneq[25] + fneq[26];
    PXY += fneq[19] - fneq[23] + fneq[20] - fneq[24] + fneq[21] - fneq[25] + fneq[22] - fneq[26];
    PXZ += fneq[19] - fneq[21] + fneq[20] - fneq[22] + fneq[23] - fneq[25] + fneq[24] - fneq[26];
    PYZ += fneq[19] - fneq[21] + fneq[20] - fneq[22] + fneq[25] - fneq[23] + fneq[26] - fneq[24];
    #endif // D3Q27
 
    d.ux[idx3] = ux_val; d.uy[idx3] = uy_val; d.uz[idx3] = uz_val;

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        float feq = gpuComputeEquilibriaSecondOrder(rho_val,ux_val,uy_val,uz_val,uu,Q);
        float he_force = COEFF_HE * feq * ( (CIX[Q] - ux_val) * ffx_val +
                                            (CIY[Q] - uy_val) * ffy_val +
                                            (CIZ[Q] - uz_val) * ffz_val ) * inv_rho_cssq;
        float fneq_scalar = (W[Q] * 4.5f) * ((CIX[Q]*CIX[Q] - CSSQ) * PXX +
                                             (CIY[Q]*CIY[Q] - CSSQ) * PYY +
                                             (CIZ[Q]*CIZ[Q] - CSSQ) * PZZ +
                                             2.0f * CIX[Q] * CIY[Q] * PXY +
                                             2.0f * CIX[Q] * CIZ[Q] * PXZ +
                                             2.0f * CIY[Q] * CIZ[Q] * PYZ);
        const int streamed_idx4 = gpuIdxGlobal4(xx,yy,zz,Q);
        d.f[streamed_idx4] = feq + (1.0f - OMEGA) * fneq_scalar + he_force; 
    }
}

__global__ void gpuEvolvePhaseField(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const int idx3 = gpuIdxGlobal3(x,y,z);

    float ux_val = d.ux[idx3];
    float uy_val = d.uy[idx3];
    float uz_val = d.uz[idx3];
    float phi_val = d.phi[idx3];
    float normx_val = d.normx[idx3]; 
    float normy_val = d.normy[idx3];
    float normz_val = d.normz[idx3];

    float phi_norm = GAMMA * phi_val * (1.0f - phi_val);
    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        const int streamed_idx4 = gpuIdxGlobal4(xx,yy,zz,Q);
        float geq = gpuComputeEquilibriaFirstOrder(phi_val,ux_val,uy_val,uz_val,Q);
        float anti_diff = W[Q] * phi_norm * (CIX[Q] * normx_val + CIY[Q] * normy_val + CIZ[Q] * normz_val);
        d.g[streamed_idx4] = geq + anti_diff;
    }
}


