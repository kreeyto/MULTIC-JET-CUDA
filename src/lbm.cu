#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "globalFunctions.cuh"

__global__ void initDist(float * __restrict__ f) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ) return;

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int idx4D = gpuIdxGlobal4(x,y,z,Q);
        f[idx4D] = W[Q];
    }
}

// =================================================================================================== //

__global__ void gpuPhaseField(
    float * __restrict__ phi,
    const float * __restrict__ g
) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const int idx = gpuIdxGlobal3(x,y,z);

    float phiVal = 0.0f;
    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int idx4D = gpuIdxGlobal4(x,y,z,Q);
        phiVal += __ldg(&g[idx4D]);
    }

    phi[idx] = phiVal;
}

__global__ void gpuGradients(
    const float * __restrict__ phi,
    float * __restrict__ normx,
    float * __restrict__ normy,
    float * __restrict__ normz,
    float * __restrict__ indicator
) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const int idx = gpuIdxGlobal3(x,y,z);

    float gradx = 3.0f * (W[1]  * phi[IDX3D(x+1,y,z)]   - W[2]  * phi[IDX3D(x-1,y,z)]
                        + W[7]  * phi[IDX3D(x+1,y+1,z)] - W[8]  * phi[IDX3D(x-1,y-1,z)]
                        + W[9]  * phi[IDX3D(x+1,y,z+1)] - W[10] * phi[IDX3D(x-1,y,z-1)]
                        + W[13] * phi[IDX3D(x+1,y-1,z)] - W[14] * phi[IDX3D(x-1,y+1,z)]
                        + W[15] * phi[IDX3D(x+1,y,z-1)] - W[16] * phi[IDX3D(x-1,y,z+1)]);

    float grady = 3.0f * (W[3]  * phi[IDX3D(x,y+1,z)]   - W[4]  * phi[IDX3D(x,y-1,z)]
                        + W[7]  * phi[IDX3D(x+1,y+1,z)] - W[8]  * phi[IDX3D(x-1,y-1,z)]
                        + W[11] * phi[IDX3D(x,y+1,z+1)] - W[12] * phi[IDX3D(x,y-1,z-1)]
                        - W[13] * phi[IDX3D(x+1,y-1,z)] + W[14] * phi[IDX3D(x-1,y+1,z)]
                        + W[17] * phi[IDX3D(x,y+1,z-1)] - W[18] * phi[IDX3D(x,y-1,z+1)]);
    
    float gradz = 3.0f * (W[5]  * phi[IDX3D(x,y,z+1)]   - W[6]  * phi[IDX3D(x,y,z-1)]
                        + W[9]  * phi[IDX3D(x+1,y,z+1)] - W[10] * phi[IDX3D(x-1,y,z-1)]
                        + W[11] * phi[IDX3D(x,y+1,z+1)] - W[12] * phi[IDX3D(x,y-1,z-1)]
                        - W[15] * phi[IDX3D(x+1,y,z-1)] + W[16] * phi[IDX3D(x-1,y,z+1)]
                        - W[17] * phi[IDX3D(x,y+1,z-1)] + W[18] * phi[IDX3D(x,y-1,z+1)]);
    
    float gmagsq = gradx*gradx + grady*grady + gradz*gradz;
    float factor = rsqrtf(fmaxf(gmagsq, 1e-9));

    normx[idx] = gradx * factor;
    normy[idx] = grady * factor;
    normz[idx] = gradz * factor; 
    indicator[idx] = gmagsq * factor;  
}

__global__ void gpuCurvature(
    const float * __restrict__ indicator,
    const float * __restrict__ normx,
    const float * __restrict__ normy,
    const float * __restrict__ normz,
    float * __restrict__ ffx,
    float * __restrict__ ffy,
    float * __restrict__ ffz
) {
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;
    const int x = tx + blockIdx.x * blockDim.x;
    const int y = ty + blockIdx.y * blockDim.y;
    const int z = tz + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ ||
        x == 0 || x == NX-1 ||
        y == 0 || y == NY-1 ||
        z == 0 || z == NZ-1) return;

    const int lx = tx + 1;
    const int ly = ty + 1;
    const int lz = tz + 1;

    __shared__ float s_normx[BLOCK_SIZE_Z+2][BLOCK_SIZE_Y+2][BLOCK_SIZE_X+2];
    __shared__ float s_normy[BLOCK_SIZE_Z+2][BLOCK_SIZE_Y+2][BLOCK_SIZE_X+2];
    __shared__ float s_normz[BLOCK_SIZE_Z+2][BLOCK_SIZE_Y+2][BLOCK_SIZE_X+2];

    const int idx = gpuIdxGlobal3(x,y,z);
    
    // load bulk into shared
    s_normx[lz][ly][lx] = normx[idx];
    s_normy[lz][ly][lx] = normy[idx];
    s_normz[lz][ly][lx] = normz[idx];

    // load halos into shared
    if (tx == 0) {
        s_normx[lz][ly][lx-1] = normx[gpuIdxGlobal3(x-1,y,z)];
        s_normy[lz][ly][lx-1] = normy[gpuIdxGlobal3(x-1,y,z)];
        s_normz[lz][ly][lx-1] = normz[gpuIdxGlobal3(x-1,y,z)];
    }
    if (tx == BLOCK_SIZE_X-1) {
        s_normx[lz][ly][lx+1] = normx[gpuIdxGlobal3(x+1,y,z)];
        s_normy[lz][ly][lx+1] = normy[gpuIdxGlobal3(x+1,y,z)];
        s_normz[lz][ly][lx+1] = normz[gpuIdxGlobal3(x+1,y,z)];
    }
    if (ty == 0) {
        s_normx[lz][ly-1][lx] = normx[gpuIdxGlobal3(x,y-1,z)];
        s_normy[lz][ly-1][lx] = normy[gpuIdxGlobal3(x,y-1,z)];
        s_normz[lz][ly-1][lx] = normz[gpuIdxGlobal3(x,y-1,z)];
    }
    if (ty == BLOCK_SIZE_Y-1) {
        s_normx[lz][ly+1][lx] = normx[gpuIdxGlobal3(x,y+1,z)];
        s_normy[lz][ly+1][lx] = normy[gpuIdxGlobal3(x,y+1,z)];
        s_normz[lz][ly+1][lx] = normz[gpuIdxGlobal3(x,y+1,z)];
    }
    if (tz == 0) {
        s_normx[lz-1][ly][lx] = normx[gpuIdxGlobal3(x,y,z-1)];
        s_normy[lz-1][ly][lx] = normy[gpuIdxGlobal3(x,y,z-1)];
        s_normz[lz-1][ly][lx] = normz[gpuIdxGlobal3(x,y,z-1)];
    }
    if (tz == BLOCK_SIZE_Z-1) {
        s_normx[lz+1][ly][lx] = normx[gpuIdxGlobal3(x,y,z+1)];
        s_normy[lz+1][ly][lx] = normy[gpuIdxGlobal3(x,y,z+1)];
        s_normz[lz+1][ly][lx] = normz[gpuIdxGlobal3(x,y,z+1)];
    }
    __syncthreads();

    float curvature = -3.0f * (W[1]  * (CIX[1]  * s_normx[lz][ly][lx+1]   + CIY[1]  * s_normy[lz][ly][lx+1]   + CIZ[1]  * s_normz[lz][ly][lx+1])
                             + W[2]  * (CIX[2]  * s_normx[lz][ly][lx-1]   + CIY[2]  * s_normy[lz][ly][lx-1]   + CIZ[2]  * s_normz[lz][ly][lx-1])
                             + W[3]  * (CIX[3]  * s_normx[lz][ly+1][lx]   + CIY[3]  * s_normy[lz][ly+1][lx]   + CIZ[3]  * s_normz[lz][ly+1][lx])
                             + W[4]  * (CIX[4]  * s_normx[lz][ly-1][lx]   + CIY[4]  * s_normy[lz][ly-1][lx]   + CIZ[4]  * s_normz[lz][ly-1][lx])
                             + W[5]  * (CIX[5]  * s_normx[lz+1][ly][lx]   + CIY[5]  * s_normy[lz+1][ly][lx]   + CIZ[5]  * s_normz[lz+1][ly][lx])
                             + W[6]  * (CIX[6]  * s_normx[lz-1][ly][lx]   + CIY[6]  * s_normy[lz-1][ly][lx]   + CIZ[6]  * s_normz[lz-1][ly][lx])
                             + W[7]  * (CIX[7]  * s_normx[lz][ly+1][lx+1] + CIY[7]  * s_normy[lz][ly+1][lx+1] + CIZ[7]  * s_normz[lz][ly+1][lx+1])
                             + W[8]  * (CIX[8]  * s_normx[lz][ly-1][lx-1] + CIY[8]  * s_normy[lz][ly-1][lx-1] + CIZ[8]  * s_normz[lz][ly-1][lx-1])
                             + W[9]  * (CIX[9]  * s_normx[lz+1][ly][lx+1] + CIY[9]  * s_normy[lz+1][ly][lx+1] + CIZ[9]  * s_normz[lz+1][ly][lx+1])
                             + W[10] * (CIX[10] * s_normx[lz-1][ly][lx-1] + CIY[10] * s_normy[lz-1][ly][lx-1] + CIZ[10] * s_normz[lz-1][ly][lx-1])
                             + W[11] * (CIX[11] * s_normx[lz+1][ly+1][lx] + CIY[11] * s_normy[lz+1][ly+1][lx] + CIZ[11] * s_normz[lz+1][ly+1][lx])
                             + W[12] * (CIX[12] * s_normx[lz-1][ly-1][lx] + CIY[12] * s_normy[lz-1][ly-1][lx] + CIZ[12] * s_normz[lz-1][ly-1][lx])
                             + W[13] * (CIX[13] * s_normx[lz][ly-1][lx+1] + CIY[13] * s_normy[lz][ly-1][lx+1] + CIZ[13] * s_normz[lz][ly-1][lx+1])
                             + W[14] * (CIX[14] * s_normx[lz][ly+1][lx-1] + CIY[14] * s_normy[lz][ly+1][lx-1] + CIZ[14] * s_normz[lz][ly+1][lx-1])
                             + W[15] * (CIX[15] * s_normx[lz-1][ly][lx+1] + CIY[15] * s_normy[lz-1][ly][lx+1] + CIZ[15] * s_normz[lz-1][ly][lx+1])
                             + W[16] * (CIX[16] * s_normx[lz+1][ly][lx-1] + CIY[16] * s_normy[lz+1][ly][lx-1] + CIZ[16] * s_normz[lz+1][ly][lx-1])
                             + W[17] * (CIX[17] * s_normx[lz-1][ly+1][lx] + CIY[17] * s_normy[lz-1][ly+1][lx] + CIZ[17] * s_normz[lz-1][ly+1][lx])
                             + W[18] * (CIX[18] * s_normx[lz+1][ly-1][lx] + CIY[18] * s_normy[lz+1][ly-1][lx] + CIZ[18] * s_normz[lz+1][ly-1][lx]));

    float mult = SIGMA * curvature;
    float indVal = indicator[idx];
    float normxVal = s_normx[lz][ly][lx];
    float normyVal = s_normy[lz][ly][lx];
    float normzVal = s_normz[lz][ly][lx];

    ffx[idx] = mult * normxVal * indVal;
    ffy[idx] = mult * normyVal * indVal;
    ffz[idx] = mult * normzVal * indVal;
}


// =================================================================================================== //



// =================================================================================================== //

__global__ void gpuMomOneCollisionStream(
    float * __restrict__ ux,
    float * __restrict__ uy,
    float * __restrict__ uz,
    float * __restrict__ rho,
    const float * __restrict__ ffx,
    const float * __restrict__ ffy,
    const float * __restrict__ ffz,
    float * __restrict__ f
) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const int idx = gpuIdxGlobal3(x,y,z);
    
    float fneq[NLINKS];
    float fVal[NLINKS];

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int idx4D = gpuIdxGlobal4(x,y,z,Q);
        fVal[Q] = f[idx4D];
    }

    float rhoVal = fVal[0] + fVal[1] + fVal[2] + fVal[3] + fVal[4] + fVal[5] +
                   fVal[6] + fVal[7] + fVal[8] + fVal[9] + fVal[10] + fVal[11] +
                   fVal[12] + fVal[13] + fVal[14] + fVal[15] + fVal[16] + fVal[17] + fVal[18];

    float invRho = 1.0f / rhoVal;

    float sumUx = invRho * (fVal[1] - fVal[2] + fVal[7] - fVal[8] + fVal[9] - fVal[10] + fVal[13] - fVal[14] + fVal[15] - fVal[16]);
    float sumUy = invRho * (fVal[3] - fVal[4] + fVal[7] - fVal[8] + fVal[11] - fVal[12] + fVal[14] - fVal[13] + fVal[17] - fVal[18]);
    float sumUz = invRho * (fVal[5] - fVal[6] + fVal[9] - fVal[10] + fVal[11] - fVal[12] + fVal[16] - fVal[15] + fVal[18] - fVal[17]);

    float ffxVal = ffx[idx];
    float ffyVal = ffy[idx];
    float ffzVal = ffz[idx];

    float halfFx = ffxVal * 0.5f * invRho;
    float halfFy = ffyVal * 0.5f * invRho;
    float halfFz = ffzVal * 0.5f * invRho;

    float uxVal = sumUx + halfFx;
    float uyVal = sumUy + halfFy;
    float uzVal = sumUz + halfFz;

    float uu = 1.5f * (uxVal*uxVal + uyVal*uyVal + uzVal*uzVal);
    float invRhoCssq = 3.0f * invRho;

    float auxHe = 1.0f - OMEGA / 2.0f;

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        float pre = gpuFeq(rhoVal,uxVal,uyVal,uzVal,uu,Q);
        float HeF = auxHe * pre * ((CIX[Q] - uxVal) * ffxVal +
                                   (CIY[Q] - uyVal) * ffyVal +
                                   (CIZ[Q] - uzVal) * ffzVal) * invRhoCssq;
        float feq = pre - HeF; 
        fneq[Q] = fVal[Q] - feq;
    }

    float PXX = fneq[1] + fneq[2] + fneq[7] + fneq[8] + fneq[9] + fneq[10] + fneq[13] + fneq[14] + fneq[15] + fneq[16];
    float PYY = fneq[3] + fneq[4] + fneq[7] + fneq[8] + fneq[11] + fneq[12] + fneq[13] + fneq[14] + fneq[17] + fneq[18];
    float PZZ = fneq[5] + fneq[6] + fneq[9] + fneq[10] + fneq[11] + fneq[12] + fneq[15] + fneq[16] + fneq[17] + fneq[18];
    float PXY = fneq[7] - fneq[13] + fneq[8] - fneq[14];
    float PXZ = fneq[9] - fneq[15] + fneq[10] - fneq[16];
    float PYZ = fneq[11] - fneq[17] + fneq[12] - fneq[18];

    ux[idx] = uxVal; uy[idx] = uyVal; uz[idx] = uzVal;

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int xx = x + CIX[Q], yy = y + CIY[Q], zz = z + CIZ[Q];
        float feq = gpuFeq(rhoVal,uxVal,uyVal,uzVal,uu,Q);
        float HeF = auxHe * feq * ( (CIX[Q] - uxVal) * ffxVal +
                                    (CIY[Q] - uyVal) * ffyVal +
                                    (CIZ[Q] - uzVal) * ffzVal ) * invRhoCssq;
        float fneq = (W[Q] * 4.5f) * gpuTensor2(PXX,PYY,PZZ,PXY,PXZ,PYZ,Q);
        const int str = gpuIdxGlobal4(xx,yy,zz,Q);
        f[str] = feq + (1.0f - OMEGA) * fneq + HeF; 
    }
}

__global__ void gpuTwoCollisionStream(
    float * __restrict__ g,
    const float * __restrict__ ux,
    const float * __restrict__ uy,
    const float * __restrict__ uz,
    const float * __restrict__ phi,
    const float * __restrict__ normx,
    const float * __restrict__ normy,
    const float * __restrict__ normz
) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const int idx = gpuIdxGlobal3(x,y,z);

    float uxVal = ux[idx];
    float uyVal = uy[idx];
    float uzVal = uz[idx];
    float phiVal = phi[idx];
    float normxVal = normx[idx]; 
    float normyVal = normy[idx];
    float normzVal = normz[idx];
    
    float uu = 1.5f * (uxVal*uxVal + uyVal*uyVal + uzVal*uzVal);
    float phiNorm = SHARP_C * phiVal * (1.0f - phiVal);
    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int xx = x + CIX[Q], yy = y + CIY[Q], zz = z + CIZ[Q];
        float geq = gpuFeq(phiVal,uxVal,uyVal,uzVal,uu,Q);
        float Hi = W[Q] * phiNorm * (CIX[Q] * normxVal + CIY[Q] * normyVal + CIZ[Q] * normzVal);
        const int str = gpuIdxGlobal4(xx,yy,zz,Q);
        g[str] = geq + Hi;
    }
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void gpuInflow(
    float * __restrict__ rho,
    float * __restrict__ phi,
    float * __restrict__ ux,
    float * __restrict__ uy,
    float * __restrict__ uz,
    float * __restrict__ f,
    float * __restrict__ g,
    const float * __restrict__ ffx,
    const float * __restrict__ ffy,
    const float * __restrict__ ffz
) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = 0;  

    if (x >= NX || y >= NY) return;

    float cx = NX * 0.5f;
    float cy = NY * 0.5f;

    float dx = x - cx;
    float dy = y - cy;
    float Ri = sqrtf(dx*dx + dy*dy);
    
    if (Ri > DIAM) return;
    float Ri_norm = Ri / DIAM;

    float weight = 1.0f - smoothstep(0.6f, 1.0f, Ri_norm);
    float phiIn = weight;
    float uzIn = U_JET * phiIn; 
    
    const int idxIn = gpuIdxGlobal3(x,y,z);

    float ffxVal = ffx[idxIn];
    float ffyVal = ffy[idxIn];
    float ffzVal = ffz[idxIn];

    float rhoVal = 1.0f;
    float uu = 1.5f * (uzIn * uzIn);
    float auxHe = 1.0f - OMEGA / 2.0f;  

    rho[idxIn] = rhoVal;
    phi[idxIn] = phiIn;
    ux[idxIn] = 0.0f;
    uy[idxIn] = 0.0f;
    uz[idxIn] = uzIn; 

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int xx = x + CIX[Q], yy = y + CIY[Q], zz = z + CIZ[Q];
        float feq = gpuFeq(1.0f, 0.0f, 0.0f, uzIn, uu, Q);
        float HeF = auxHe * feq * (CIX[Q] * ffxVal +
                                   CIY[Q] * ffyVal +
                                  (CIZ[Q] - uzIn) * ffzVal) * 3.0f; // was * invRho
        const int str = gpuIdxGlobal4(xx, yy, zz, Q);
        f[str] = feq + HeF;
    }

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int xx = x + CIX[Q], yy = y + CIY[Q], zz = z + CIZ[Q];
        float geq = gpuFeq(phiIn, 0.0f, 0.0f, uzIn, uu, Q);
        const int str = gpuIdxGlobal4(xx, yy, zz, Q);
        g[str] = geq;
    }
}

// =================================================================================================== //

