#include "hip/hip_runtime.h"
#include "kernels.cuh"

// ================================================================================================== //

__global__ void gpuPhaseField(
    float * __restrict__ phi,
    const float * __restrict__ g,
    const int NX, const int NY, const int NZ
) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    int idx = inline3D(x,y,z,NX,NY);
    float phiVal = 0.0f;

    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {
        int idx4D = inline4D(x,y,z,Q,NX,NY,NZ);
        phiVal += g[idx4D];
    }

    phi[idx] = phiVal;
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void gpuGradients(
    const float * __restrict__ phi,
    float * __restrict__ normx,
    float * __restrict__ normy,
    float * __restrict__ normz,
    float * __restrict__ indicator,
    const int NX, const int NY, const int NZ
) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    int idx = inline3D(x,y,z,NX,NY);

    float gradx = 0.0f, grady = 0.0f, gradz = 0.0f;
    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {

        int xx = (x + CIX[Q] + NX) % NX;
        int yy = (y + CIY[Q] + NY) % NY;
        int zz = z + CIZ[Q];
        zz = min( max(zz, 1), NZ-2 );

        int offset = inline3D(xx,yy,zz,NX,NY);
        float phiVal = phi[offset];
        float coef = 3.0f * W[Q];
        gradx += coef * CIX[Q] * phiVal;
        grady += coef * CIY[Q] * phiVal;
        gradz += coef * CIZ[Q] * phiVal;
    }
    
    float gmagsq = gradx*gradx + grady*grady + gradz*gradz;
    float factor = rsqrtf(fmaxf(gmagsq, 1e-9));

    normx[idx] = gradx * factor;
    normy[idx] = grady * factor;
    normz[idx] = gradz * factor; 
    indicator[idx] = gmagsq * factor;  
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void gpuCurvature(
    const float * __restrict__ indicator,
    const float * __restrict__ normx,
    const float * __restrict__ normy,
    const float * __restrict__ normz,
    float * __restrict__ ffx,
    float * __restrict__ ffy,
    float * __restrict__ ffz,
    const int NX, const int NY, const int NZ
) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    int idx = inline3D(x,y,z,NX,NY);

    float normxVal = normx[idx];
    float normyVal = normy[idx];
    float normzVal = normz[idx];
    float indVal = indicator[idx];
    float curvature = 0.0f;

    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {

        int xx = (x + CIX[Q] + NX) % NX;
        int yy = (y + CIY[Q] + NY) % NY;
        int zz = z + CIZ[Q];
        zz = min( max(zz, 1), NZ-2 );

        int offset = inline3D(xx,yy,zz,NX,NY);
        float nox = normx[offset];
        float noy = normy[offset];
        float noz = normz[offset];
        float coef = 3.0f * W[Q];
        curvature -= coef * (CIX[Q]*nox + CIY[Q]*noy + CIZ[Q]*noz);
    }

    float mult = SIGMA * curvature;

    ffx[idx] = mult * normxVal * indVal;
    ffy[idx] = mult * normyVal * indVal;
    ffz[idx] = mult * normzVal * indVal;
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void gpuMomCollisionStream(
    float * __restrict__ ux,
    float * __restrict__ uy,
    float * __restrict__ uz,
    float * __restrict__ rho,
    const float * __restrict__ ffx,
    const float * __restrict__ ffy,
    const float * __restrict__ ffz,
    float * __restrict__ f,
    const int NX, const int NY, const int NZ
) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    int idx = inline3D(x,y,z,NX,NY);
    
    float fneq[NLINKS];
    float fVal[NLINKS];

    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {
        int idx4D = inline4D(x,y,z,Q,NX,NY,NZ);
        fVal[Q] = f[idx4D];
    }

    float rhoVal = 0.0f;
    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) 
        rhoVal += fVal[Q];

    float invRho = 1.0f / rhoVal;

    float sumUx = invRho * (fVal[1] - fVal[2] + fVal[7] - fVal[8] + fVal[9] - fVal[10] + fVal[13] - fVal[14] + fVal[15] - fVal[16]);
    float sumUy = invRho * (fVal[3] - fVal[4] + fVal[7] - fVal[8] + fVal[11] - fVal[12] + fVal[14] - fVal[13] + fVal[17] - fVal[18]);
    float sumUz = invRho * (fVal[5] - fVal[6] + fVal[9] - fVal[10] + fVal[11] - fVal[12] + fVal[16] - fVal[15] + fVal[18] - fVal[17]);

    float ffxVal = ffx[idx];
    float ffyVal = ffy[idx];
    float ffzVal = ffz[idx];

    // the factor 1/2 here emerges from he et al. forcing scheme, where:
    // u = 1/rho * sum_i(c_i*f_i) + A * F/rho
    // thus, with A = 1/2:
    // u = 1/rho * sum_i(c_i*f_i) + F/(2*rho)
    float halfFx = ffxVal * 0.5f * invRho;
    float halfFy = ffyVal * 0.5f * invRho;
    float halfFz = ffzVal * 0.5f * invRho;

    float uxVal = sumUx + halfFx;
    float uyVal = sumUy + halfFy;
    float uzVal = sumUz + halfFz;

    float uu = 1.5f * (uxVal*uxVal + uyVal*uyVal + uzVal*uzVal);
    float invRhoCssq = 3.0f / rhoVal;

    float auxHe = 1.0f - OMEGA / 2.0f;

    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {
        float cu = 3.0f * (uxVal * CIX[Q] + uyVal * CIY[Q] + uzVal * CIZ[Q]);
        float eqBase = rhoVal * (cu + 0.5f * cu*cu - uu);
        float common = W[Q] * (rhoVal + eqBase);
        float HeF = auxHe * common * ((CIX[Q] - uxVal) * ffxVal +
                                      (CIY[Q] - uyVal) * ffyVal +
                                      (CIZ[Q] - uzVal) * ffzVal) * invRhoCssq;
        float feq = common - HeF; 
        fneq[Q] = fVal[Q] - feq;
    }

    float PXX = fneq[1] + fneq[2] + fneq[7] + fneq[8] + fneq[9] + fneq[10] + fneq[13] + fneq[14] + fneq[15] + fneq[16];
    float PYY = fneq[3] + fneq[4] + fneq[7] + fneq[8] + fneq[11] + fneq[12] + fneq[13] + fneq[14] + fneq[17] + fneq[18];
    float PZZ = fneq[5] + fneq[6] + fneq[9] + fneq[10] + fneq[11] + fneq[12] + fneq[15] + fneq[16] + fneq[17] + fneq[18];
    float PXY = fneq[7] - fneq[13] + fneq[8] - fneq[14];
    float PXZ = fneq[9] - fneq[15] + fneq[10] - fneq[16];
    float PYZ = fneq[11] - fneq[17] + fneq[12] - fneq[18];

    ux[idx] = uxVal; uy[idx] = uyVal; uz[idx] = uzVal;

    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {

        int xx = (x + CIX[Q] + NX) % NX;
        int yy = (y + CIY[Q] + NY) % NY;
        int zz = z + CIZ[Q];
        zz = min( max(zz, 1), NZ-2 );

        float cu = 3.0f * (uxVal*CIX[Q] + uyVal*CIY[Q] + uzVal*CIZ[Q]);
        float feq = W[Q] * (rhoVal + rhoVal * (cu + 0.5f * cu*cu - uu));
        float HeF = auxHe * feq * ( (CIX[Q] - uxVal) * ffxVal +
                                    (CIY[Q] - uyVal) * ffyVal +
                                    (CIZ[Q] - uzVal) * ffzVal ) * invRhoCssq;
        float fneq = (W[Q] * 4.5f) * ((CIX[Q]*CIX[Q] - CSSQ) * PXX +
                                      (CIY[Q]*CIY[Q] - CSSQ) * PYY +
                                      (CIZ[Q]*CIZ[Q] - CSSQ) * PZZ +
                                       2.0f * CIX[Q] * CIY[Q] * PXY +
                                       2.0f * CIX[Q] * CIZ[Q] * PXZ +
                                       2.0f * CIY[Q] * CIZ[Q] * PYZ
                                     );
        int offset = inline4D(xx,yy,zz,Q,NX,NY,NZ);
        f[offset] = feq + (1.0f - OMEGA) * fneq + HeF; 
    }
}

__global__ void gpuPhaseCollisionStream(
    float * __restrict__ g,
    const float * __restrict__ ux,
    const float * __restrict__ uy,
    const float * __restrict__ uz,
    const float * __restrict__ phi,
    const float * __restrict__ normx,
    const float * __restrict__ normy,
    const float * __restrict__ normz,
    const int NX, const int NY, const int NZ
) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    int idx = inline3D(x,y,z,NX,NY);

    float uxVal = ux[idx];
    float uyVal = uy[idx];
    float uzVal = uz[idx];
    float phiVal = phi[idx];
    float normxVal = normx[idx]; 
    float normyVal = normy[idx];
    float normzVal = normz[idx];
    
    float uu = 1.5f * (uxVal*uxVal + uyVal*uyVal + uzVal*uzVal);
    float phiNorm = SHARP_C * phiVal * (1.0f - phiVal);
    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {

        int xx = (x + CIX[Q] + NX) % NX;
        int yy = (y + CIY[Q] + NY) % NY;
        int zz = z + CIZ[Q];
        zz = min( max(zz, 1), NZ-2 );

        float cu = 3.0f * (uxVal * CIX[Q] + uyVal * CIY[Q] + uzVal * CIZ[Q]);
        // was using first order
        //float geq = W[Q] * phiVal * (1.0f + cu);
        float geq = W[Q] * (phiVal + phiVal * (cu + 0.5f * cu*cu - uu));
        float Hi = W[Q] * phiNorm * (CIX[Q] * normxVal + CIY[Q] * normyVal + CIZ[Q] * normzVal);
        int offset = inline4D(xx,yy,zz,Q,NX,NY,NZ);
        g[offset] = geq + Hi;
    }
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void gpuInflow(
    float * __restrict__ rho,
    float * __restrict__ phi,
    float * __restrict__ ux,
    float * __restrict__ uy,
    float * __restrict__ uz,
    float * __restrict__ f,
    float * __restrict__ g,
    const float * __restrict__ ffx,
    const float * __restrict__ ffy,
    const float * __restrict__ ffz,
    const float U_JET, const int DIAM,
    const int NX, const int NY, const int NZ
    //const int STEP, const int MACRO_SAVE
) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (z != 0) return; 
    
    float cx = NX * 0.5f;
    float cy = NY * 0.5f;

    float dx = x - cx;
    float dy = y - cy;
    float Ri = sqrtf(dx*dx + dy*dy);
    
    if (Ri > DIAM) return;

    float phiIn = 1.0f; //0.5f + 0.5f * tanh(2.0f * (DIAM - Ri) / INTERFACE_WIDTH);
    float uzIn = U_JET * phiIn; 
    
    int idxIn = inline3D(x,y,z,NX,NY);

    float ffxVal = ffx[idxIn];
    float ffyVal = ffy[idxIn];
    float ffzVal = ffz[idxIn];

    float rhoVal = 1.0f;
    float uu = 1.5f * (uzIn * uzIn);
    float invRhoCssq = 3.0f / rhoVal;
    float auxHe = 1.0f - OMEGA / 2.0f;  

    rho[idxIn] = rhoVal;
    phi[idxIn] = phiIn;
    ux[idxIn] = 0.0f;
    uy[idxIn] = 0.0f;
    uz[idxIn] = uzIn; 

    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {
        float cu = 3.0f * uzIn * CIZ[Q];
        float feq = W[Q] * (1.0f + (cu + 0.5f * cu*cu - uu));
        float HeF = auxHe * feq * (CIX[Q] * ffxVal +
                                   CIY[Q] * ffyVal +
                                   (CIZ[Q] - uzIn) * ffzVal) * invRhoCssq;

        int xx = (x + CIX[Q] + NX) % NX;
        int yy = (y + CIY[Q] + NY) % NY;
        int zz = z + CIZ[Q];
        zz = min( max(zz, 1), NZ-2 );
        
        int offset = inline4D(xx,yy,zz,Q,NX,NY,NZ);
        f[offset] = feq + HeF;
    }

    #pragma unroll 19
    for (int Q = 0; Q < NLINKS; ++Q) {
        float cu = 3.0f * uzIn * CIZ[Q];
        // was using first order
        //float geq = W[Q] * phiIn * (1.0f + cu);
        float geq = W[Q] * (phiIn + phiIn * (cu + 0.5f * cu*cu - uu));

        int xx = (x + CIX[Q] + NX) % NX;
        int yy = (y + CIY[Q] + NY) % NY;
        int zz = z + CIZ[Q];
        zz = min( max(zz, 1), NZ-2 );

        int offset = inline4D(xx,yy,zz,Q,NX,NY,NZ);
        g[offset] = geq;
    }
}

// =================================================================================================== //

