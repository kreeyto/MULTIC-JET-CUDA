#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuApplyOutflow(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = NZ - 1;
    if (x >= NX || y >= NY) return;

    const int idx3_out = gpuIdxGlobal3(x,y,z);
    const int idx3_prev = gpuIdxGlobal3(x,y,z-1);

    d.rho[idx3_out] = d.rho[idx3_prev];
    d.ux[idx3_out]  = d.ux[idx3_prev];
    d.uy[idx3_out]  = d.uy[idx3_prev];
    d.uz[idx3_out]  = d.uz[idx3_prev];
    d.phi[idx3_out] = d.phi[idx3_prev];

    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int idx4_out = gpuIdxGlobal4(x,y,z,Q);
        const int idx4_prev = gpuIdxGlobal4(x,y,z-1,Q);
        d.f[idx4_out] = d.f[idx4_prev];
    }
    
    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int idx4_out = gpuIdxGlobal4(x,y,z,Q);
        const int idx4_prev = gpuIdxGlobal4(x,y,z-1,Q);
        d.g[idx4_out] = d.g[idx4_prev];
    }
}

__global__ void gpuApplyInflow(LBMFields d, const int STEP) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = 0;  

    if (x >= NX || y >= NY) return;

    float center_x = 0.5f * NX;
    float center_y = 0.5f * NY;

    float dx = x-center_x, dy = y-center_y;
    float radial_dist = sqrtf(dx*dx + dy*dy);
    
    float radius = 0.5f * DIAM;
    if (radial_dist > radius) return;

    float radial_dist_norm = radial_dist / radius;

    //float phi_in = 0.5f + 0.5f * tanhf(2.0f * (radius - radial_dist) / 3.0f);
    float smoothing_factor = 1.0f - gpuSmoothstep(0.6f,1.0f,radial_dist_norm);
    float phi_in = smoothing_factor;

    #ifdef PERTURBATION
        float uz_in = U_JET * (1.0f + DATAZ[STEP/MACRO_SAVE] * 1000) * phi_in; 
    #else
        float uz_in = U_JET * phi_in; 
    #endif
    
    const int idx3_in = gpuIdxGlobal3(x,y,z);

    float ffx_val = d.ffx[idx3_in];
    float ffy_val = d.ffy[idx3_in];
    float ffz_val = d.ffz[idx3_in];

    float rho_val = 1.0f;
    float uu = 1.5f * (uz_in * uz_in);

    d.rho[idx3_in] = rho_val;
    d.phi[idx3_in] = phi_in;
    d.ux[idx3_in] = 0.0f;
    d.uy[idx3_in] = 0.0f;
    d.uz[idx3_in] = uz_in; 

    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        float feq = gpuComputeEquilibriaSecondOrder(rho_val,0.0f,0.0f,uz_in,uu,Q);
        float he_force = COEFF_HE * feq * (CIX[Q] * ffx_val +
                                           CIY[Q] * ffy_val +
                                          (CIZ[Q] - uz_in) * ffz_val) * 3.0f; 
        const int streamed_idx4 = gpuIdxGlobal4(xx,yy,zz,Q);
        d.f[streamed_idx4] = feq + he_force;
    }

    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int xx = x + CIX_G[Q];
        const int yy = y + CIY_G[Q];
        const int zz = z + CIZ_G[Q];
        float geq = gpuComputeEquilibriaFirstOrder(phi_in,0.0f,0.0f,uz_in,Q);
        const int streamed_idx4 = gpuIdxGlobal4(xx,yy,zz,Q);
        d.g[streamed_idx4] = geq;
    }
}

