#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuInitDistributions(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ) return;

    #pragma unroll NLINKS
    for (int Q = 0; Q < NLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        d.f[idx4] = W[Q];
    }
}

__constant__ float CSSQ;
__constant__ float OMEGA;
__constant__ float GAMMA;
__constant__ float SIGMA;
__constant__ float COEFF_HE;

__constant__ float W[NLINKS];
__constant__ int CIX[NLINKS], CIY[NLINKS], CIZ[NLINKS];

#ifdef PERTURBATION
    __constant__ float DATAZ[200];
#endif

LBMFields lbm;
                                         
// =============================================================================================================================================================== //

void initDeviceVars() {
    size_t SIZE = NX * NY * NZ * sizeof(float);            
    size_t DIST_SIZE = NX * NY * NZ * NLINKS * sizeof(float); 

    checkCudaErrors(hipMalloc(&lbm.phi, SIZE));
    checkCudaErrors(hipMalloc(&lbm.rho, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ux, SIZE));
    checkCudaErrors(hipMalloc(&lbm.uy, SIZE));
    checkCudaErrors(hipMalloc(&lbm.uz, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normx, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normy, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normz, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ind, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffx, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffy, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffz, SIZE));
    checkCudaErrors(hipMalloc(&lbm.f, DIST_SIZE));
    checkCudaErrors(hipMalloc(&lbm.g, DIST_SIZE));

    /*
    checkCudaErrors(hipMemset(lbm.ux, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.uy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.uz, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.phi, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normx, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normz, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffx, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffz, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.g, 0, DIST_SIZE));
    */

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CSSQ), &H_CSSQ, sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(OMEGA), &H_OMEGA, sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(GAMMA), &H_GAMMA, sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(SIGMA), &H_SIGMA, sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(COEFF_HE), &H_COEFF_HE, sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W), &H_W, NLINKS * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIX), &H_CIX, NLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIY), &H_CIY, NLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIZ), &H_CIZ, NLINKS * sizeof(int)));

    #ifdef PERTURBATION
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DATAZ), &H_DATAZ, 200 * sizeof(float)));
    #endif

    getLastCudaError("initDeviceVars: post-initialization");
}

