#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuInitDistributions(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ) return;

    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        d.f[idx4] = W[Q];
    }
}

__constant__ float CSSQ;
__constant__ float OMEGA;
__constant__ float GAMMA;
__constant__ float SIGMA;
__constant__ float COEFF_HE;

__constant__ float W[FLINKS];
__constant__ float W_G[GLINKS];

__constant__ int CIX[FLINKS], CIY[FLINKS], CIZ[FLINKS];

#ifdef PERTURBATION
    __constant__ float DATAZ[200];
#endif

LBMFields lbm;
                                         
// =============================================================================================================================================================== //

void initDeviceVars() {
    size_t SIZE =        NX * NY * NZ          * sizeof(float);            
    size_t F_DIST_SIZE = NX * NY * NZ * FLINKS * sizeof(float); 
    size_t G_DIST_SIZE = NX * NY * NZ * GLINKS * sizeof(float); 

    checkCudaErrors(hipMalloc(&lbm.phi,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.rho,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ux,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uy,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uz,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.normx, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normy, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normz, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ind,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffx,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffy,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffz,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.f,     F_DIST_SIZE));
    checkCudaErrors(hipMalloc(&lbm.g,     G_DIST_SIZE));

    /*
    checkCudaErrors(hipMemset(lbm.ux, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.uy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.uz, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.phi, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normx, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normz, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffx, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffz, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.g, 0, G_DIST_SIZE));
    */

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CSSQ),     &H_CSSQ,     sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(OMEGA),    &H_OMEGA,    sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(GAMMA),    &H_GAMMA,    sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(SIGMA),    &H_SIGMA,    sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(COEFF_HE), &H_COEFF_HE, sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W),   &H_W,   FLINKS * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W_G), &H_W_G, GLINKS * sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIX),   &H_CIX,   FLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIY),   &H_CIY,   FLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIZ),   &H_CIZ,   FLINKS * sizeof(int)));

    #ifdef PERTURBATION
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DATAZ), &H_DATAZ, 200 * sizeof(float)));
    #endif

    getLastCudaError("initDeviceVars: post-initialization");
}

