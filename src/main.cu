#include "kernels.cuh"
#include "auxFunctions.cuh"
#include "var.cuh"

int main(int argc, char* argv[]) {
    auto start_time = chrono::high_resolution_clock::now();
    if (argc < 4) {
        cerr << "Erro: Uso: " << argv[0] << " F<fluid velocity set> P<phase field velocity set> <id>" << endl;
        return 1;
    }
    string fluid_model = argv[1];
    string phase_model = argv[2];
    string id = argv[3];

    string base_dir;   
    #ifdef _WIN32
        base_dir = "..\\";
    #else
        base_dir = "../";
    #endif
    string model_dir = base_dir + "bin/" + fluid_model + "_" + phase_model + "/";
    string sim_dir = model_dir + id + "/";
    #ifdef _WIN32
        string mkdir_command = "mkdir \"" + sim_dir + "\"";
    #else
        string mkdir_command = "mkdir -p \"" + sim_dir + "\"";
    #endif
    int ret = system(mkdir_command.c_str());
    (void)ret;

    // ============================================================================================================================================================= //

    // ========================= //
    int stamp = 100, nsteps = 25000;
    // ========================= //
    initializeVars();

    string info_file = sim_dir + id + "_info.txt";
    float h_tau;
    hipMemcpyFromSymbol(&h_tau, HIP_SYMBOL(TAU), sizeof(float), 0, hipMemcpyDeviceToHost);
    generateSimulationInfoFile(info_file, nx, ny, nz, stamp, nsteps, h_tau, id, fluid_model);

    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // STREAMS
    hipStream_t mainStream, collFluid, collPhase;
    hipStreamCreate(&mainStream);
    hipStreamCreate(&collFluid);
    hipStreamCreate(&collPhase);

    // ================== INIT ================== //

        initTensor<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
            d_pxx, d_pyy, d_pzz, 
            d_pxy, d_pxz, d_pyz,
            d_rho, nx, ny, nz
        );

        initPhase<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
            d_phi, d_half, nx, ny, nz
        ); 

        initDist<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
            d_rho, d_phi, d_f, d_g, nx, ny, nz
        ); 

    // ========================================= //

    vector<float> phi_host(nx * ny * nz);

    for (int t = 0; t <= nsteps ; ++t) {
        cout << "Passo " << t << " de " << nsteps << " iniciado..." << endl;

        // ================= PHASE FIELD ================= //

            phiCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_phi, d_g, nx, ny, nz
            ); 

        // =============================================== // 
        


        // ===================== NORMALS ===================== //

            gradCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_phi, d_normx, d_normy, d_normz, 
                d_indicator, 
                nx, ny, nz
            ); 

        // =================================================== // 



        // ==================== CURVATURE ==================== //

            curvatureCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_curvature, d_indicator,
                d_normx, d_normy, d_normz, 
                d_ffx, d_ffy, d_ffz,
                nx, ny, nz
            ); 

        // =================================================== //   


        
        // ===================== MOMENTI ===================== //

            momentiCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_ux, d_uy, d_uz, d_rho,
                d_ffx, d_ffy, d_ffz, d_f,
                d_pxx, d_pyy, d_pzz,
                d_pxy, d_pxz, d_pyz,
                nx, ny, nz
            ); 

        // ================================================== //   

        

        // ==================== COLLISION & STREAMING ==================== //
            
            collisionFluid<<<numBlocks, threadsPerBlock, 0, collFluid>>> (
                d_f, d_ux, d_uy, d_uz, 
                d_ffx, d_ffy, d_ffz, d_rho,
                d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz, 
                nx, ny, nz
            ); 

            collisionPhase<<<numBlocks, threadsPerBlock, 0, collPhase>>> (
                d_g, d_ux, d_uy, d_uz, 
                d_phi, d_normx, d_normy, d_normz, 
                nx, ny, nz
            ); 

            hipStreamSynchronize(collFluid);
            hipStreamSynchronize(collPhase);
        // =============================================================== //    


    
        // ========================================== DISTRIBUTION ========================================== //

            fgBoundary<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_rho, d_phi,
                d_ux, d_uy, d_uz, d_f, d_g, 
                d_normx, d_normy, d_normz,
                d_ffx, d_ffy, d_ffz,
                u_max, d_half,
                nx, ny, nz,
                t, stamp
            ); 

        // ================================================================================================= //

        hipDeviceSynchronize();

        if (t % stamp == 0) {

            copyAndSaveToBinary(d_phi, nx * ny * nz, sim_dir, id, t, "phi");

            cout << "Passo " << t << ": Dados salvos em " << sim_dir << endl;
        }
    }

    hipStreamDestroy(mainStream);

    float *pointers[] = {d_f, d_g, d_phi, d_rho, 
                          d_normx, d_normy, d_normz, d_indicator,
                          d_curvature, d_ffx, d_ffy, d_ffz, d_ux, d_uy, d_uz,
                          d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz
                        };
    freeMemory(pointers, 21);  

    auto end_time = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_time = end_time - start_time;
    long long totalcells = static_cast<long long>(nx) * ny * nz * nsteps;
    double mlups = static_cast<double>(totalcells) / (elapsed_time.count() * 1e6);

    cout << "\n// =============================================== //\n";
    cout << "     Tempo total de execução: " << elapsed_time.count() << " segundos\n";
    cout << "     Desempenho: " << mlups << " MLUPS\n";
    cout << "// =============================================== //\n" << endl;

    return 0;
}
