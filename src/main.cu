#include "kernels.cuh"
#include "host_functions.cuh"

int main(int argc, char* argv[]) {
    if (argc < 3) {
        cerr << "Error: Usage: " << argv[0] << " <velocity set> <ID>" << endl;
        return 1;
    }
    string VELOCITY_SET = argv[1];
    string SIM_ID = argv[2];

    string SIM_DIR = createSimulationDirectory(VELOCITY_SET,SIM_ID);
    computeAndPrintOccupancy();
    initDeviceVars();

    // ================================================================================================== //

    dim3 threadsPerBlock(BLOCK_SIZE_X,BLOCK_SIZE_Y,BLOCK_SIZE_Z);
    dim3 numBlocks((NX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (NY + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (NZ + threadsPerBlock.z - 1) / threadsPerBlock.z);

    dim3 threadsPerBlockBC(BLOCK_SIZE_X*2,BLOCK_SIZE_Y*2);  
    dim3 numBlocksBC((NX + threadsPerBlockBC.x - 1) / threadsPerBlockBC.x,
                     (NY + threadsPerBlockBC.y - 1) / threadsPerBlockBC.y);    

    hipStream_t mainStream;
    checkCudaErrors(hipStreamCreate(&mainStream));

    gpuInitDistributions<<<numBlocks,threadsPerBlock,0,mainStream>>> (lbm); 
    getLastCudaError("gpuInitDistributions");

    vector<float> phi_host(NX * NY * NZ); 
    //vector<float> ux_host(NX * NY * NZ);
    //vector<float> uy_host(NX * NY * NZ);
    vector<float> uz_host(NX * NY * NZ);

    auto START_TIME = chrono::high_resolution_clock::now();
    for (int STEP = 0; STEP <= NSTEPS ; ++STEP) {
        cout << "Passo " << STEP << " de " << NSTEPS << " iniciado..." << endl;

        // ======================= INTERFACE ======================= //

            gpuComputePhaseField<<<numBlocks,threadsPerBlock,0,mainStream>>> (lbm); 
            getLastCudaError("gpuComputePhaseField");

            gpuComputeGradients<<<numBlocks,threadsPerBlock,0,mainStream>>> (lbm); 
            getLastCudaError("gpuComputeGradients");
            gpuComputeCurvature<<<numBlocks,threadsPerBlock,0,mainStream>>> (lbm); 
            getLastCudaError("gpuComputeCurvature");

        // ======================================================== // 

        

        // ==================== COLLISION & STREAMING ==================== //
            
            gpuFusedCollisionStream<<<numBlocks,threadsPerBlock,0,mainStream>>> (lbm); 
            getLastCudaError("gpuFusedCollisionStream");

            gpuEvolvePhaseField<<<numBlocks,threadsPerBlock,0,mainStream>>> (lbm); 
            getLastCudaError("gpuEvolvePhaseField");

        // =============================================================== //    


    
        // ========================================== BOUNDARY ========================================== //

            gpuApplyInflow<<<numBlocksBC,threadsPerBlockBC,0,mainStream>>> (lbm,STEP); 
            getLastCudaError("gpuApplyInflow");

        // ============================================================================================= //

        checkCudaErrors(hipDeviceSynchronize());

        if (STEP % MACRO_SAVE == 0) {

            copyAndSaveToBinary(lbm.phi, NX * NY * NZ, SIM_DIR, SIM_ID, STEP, "phi");
            //copyAndSaveToBinary(lbm.ux, NX * NY * NZ, SIM_DIR, SIM_ID, STEP, "ux");
            //copyAndSaveToBinary(lbm.uy, NX * NY * NZ, SIM_DIR, SIM_ID, STEP, "uy");
            copyAndSaveToBinary(lbm.uz, NX * NY * NZ, SIM_DIR, SIM_ID, STEP, "uz");

            cout << "Passo " << STEP << ": Dados salvos em " << SIM_DIR << endl;
        }
    }
    auto END_TIME = chrono::high_resolution_clock::now();

    checkCudaErrors(hipStreamDestroy(mainStream));
    hipFree(lbm.f); hipFree(lbm.g);
    hipFree(lbm.phi); hipFree(lbm.rho);
    hipFree(lbm.ux); hipFree(lbm.uy); hipFree(lbm.uz);
    hipFree(lbm.normx); hipFree(lbm.normy); hipFree(lbm.normz);
    hipFree(lbm.ffx); hipFree(lbm.ffy); hipFree(lbm.ffz); hipFree(lbm.ind);

    chrono::duration<double> ELAPSED_TIME = END_TIME - START_TIME;
    long long TOTAL_CELLS = static_cast<long long>(NX) * NY * NZ * NSTEPS;
    double MLUPS = static_cast<double>(TOTAL_CELLS) / (ELAPSED_TIME.count() * 1e6);

    cout << "\n// =============================================== //\n";
    cout << "     Total execution time    : " << ELAPSED_TIME.count() << " seconds\n";
    cout << "     Performance             : " << MLUPS << " MLUPS\n";
    cout << "// =============================================== //\n" << endl;

    generateSimulationInfoFile(SIM_DIR,SIM_ID,VELOCITY_SET,NSTEPS,MACRO_SAVE,H_TAU,MLUPS);
    getLastCudaError("Final sync");
    return 0;
}
