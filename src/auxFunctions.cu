#include "auxFunctions.cuh"
#include "var.cuh"

void freeMemory(float **pointers, int count) {
    for (int i = 0; i < count; ++i) {
        if (pointers[i] != nullptr) {
            hipFree(pointers[i]);
        }
    }
}

void generateSimulationInfoFile(
    const string& filepath, const int nx, const int ny, const int nz, const int stamp, const int nsteps, const float tau, 
    const string& sim_id, const string& fluid_model
) {
    try {
        ofstream file(filepath);

        if (!file.is_open()) {
            cerr << "Erro ao abrir o arquivo: " << filepath << endl;
            return;
        }

        file << "---------------------------- SIMULATION INFORMATION ----------------------------\n"
             << "                           Simulation ID: " << sim_id << '\n'
             << "                           Velocity set: " << fluid_model << '\n'
             << "                           Precision: " << PRECISION_TYPE << '\n'
             << "                           NX: " << nx << '\n'
             << "                           NY: " << ny << '\n'
             << "                           NZ: " << nz << '\n'
             << "                           NZ_TOTAL: " << nz << '\n'
             << "                           Tau: " << tau << '\n'
             << "                           Umax: 0.000000e+00\n"
             << "                           FX: 0.000000e+00\n"
             << "                           FY: 0.000000e+00\n"
             << "                           FZ: 0.000000e+00\n"
             << "                           Save steps: " << stamp << '\n'
             << "                           Nsteps: " << nsteps << '\n'
             << "                           MLUPS: 1.187970e+01\n"
             << "--------------------------------------------------------------------------------\n";

        file.close();
        cout << "Arquivo de informações da simulação criado em: " << filepath << endl;
    } catch (const exception& e) {
        cerr << "Erro ao gerar o arquivo de informações: " << e.what() << endl;
    }
}

void copyAndSaveToBinary(
    const float* d_data, size_t size, const string& sim_dir, 
    const string& id, int t, const string& var_name
) {
    vector<float> host_data(size);
    
    hipMemcpy(host_data.data(), d_data, size * sizeof(float), hipMemcpyDeviceToHost);
    
    ostringstream filename;
    filename << sim_dir << id << "_" << var_name << setw(6) << setfill('0') << t << ".bin";
    
    ofstream file(filename.str(), ios::binary);
    if (!file) {
        cerr << "Erro ao abrir o arquivo " << filename.str() << " para escrita." << endl;
        return;
    }
    file.write(reinterpret_cast<const char*>(host_data.data()), host_data.size() * sizeof(float));
    file.close();
}
