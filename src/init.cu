#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void initTensor(
    float * __restrict__ pxx,
    float * __restrict__ pyy,
    float * __restrict__ pzz,
    float * __restrict__ pxy,
    float * __restrict__ pxz,
    float * __restrict__ pyz,
    float * __restrict__ rho,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    float val = 1.0;
    pxx[idx3D] = val; pyy[idx3D] = val; pzz[idx3D] = val;
    pxy[idx3D] = val; pxz[idx3D] = val; pyz[idx3D] = val;
    rho[idx3D] = val;
}

__global__ void initPhase(
    float * __restrict__ phi, 
    int d_half, int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int inlet_pos_x = nx / 2.0;
    int inlet_pos_y = ny / 2.0;
    int inlet_size = d_half;

    if (k == 0 && 
        (i >= inlet_pos_x - inlet_size && i <= inlet_pos_x + inlet_size) &&
        (j >= inlet_pos_y - inlet_size && j <= inlet_pos_y + inlet_size))     
    {
        int idx = inline3D(i,j,k,nx,ny);
        phi[idx] = 1.0;
    }
}

// =================================================================================================== //

__global__ void initDist(
    const float * __restrict__ rho, 
    const float * __restrict__ phi, 
    float * __restrict__ f,
    float * __restrict__ g,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    float rho_val = rho[idx3D];
    float phi_val = phi[idx3D];

    for (int l = 0; l < FPOINTS; ++l) {
        int idx4D = inline4D(i,j,k,l,nx,ny,nz);
        f[idx4D] = W[l] * rho_val;
    }

    for (int l = 0; l < GPOINTS; ++l) {
        int idx4D = inline4D(i,j,k,l,nx,ny,nz);
        g[idx4D] = W_G[l] * phi_val;
    }
}